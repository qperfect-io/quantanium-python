#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "quantanium/impl/gates/device_utils.cuh"

extern "C" __global__ void __qt_cuda_stub() {}

/// Force host reference to device code so that nvlink keeps fatbinData
void __qt_force_link_gpu_symbols() {
    /// any call is enough; it will be removed by the optimizer
    if (false) {
        quantanium::square_cplx_d(nullptr, 0, nullptr);
        quantanium::scale_cplx_d(nullptr, 0.0, 0, nullptr);
        quantanium::square_cplx_f(nullptr, 0, nullptr);
        quantanium::scale_cplx_f(nullptr, 0.0f, 0, nullptr);
    }
}

